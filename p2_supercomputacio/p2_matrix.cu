
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h> 

__global__ void fill_matrix_device(int *m, int width) 
{ 
    int tx=threadIdx.x; 
    int ty=threadIdx.y; 
    
    int value=(tx+1)*(ty+1); 
    m[tx*width+ty] = value; 
}

void fill_matrix_host(int *m, int width) 
{ 
    for(int x=0;x<width;++x) { 
        for(int y=0;y<width;++y) { 
            int value=(x+1)*(y+1); 
            m[x*width+y] = value; 
        } 
    } 
} 

int main(void) 
{ 
    int width=2; 
    int size=width*width*sizeof(int); 

    int *m; 
    m = (int *)malloc(size); 
 
    fill_matrix_host(m, width); 
    
    int *dev_m; 
    hipMalloc((void **)&dev_m, size); 
    dim3 dimGrid(1, 1); 
    dim3 dimBlock(width, width); 
    
    fill_matrix_device<<<dimGrid, dimBlock>>>(dev_m, width); 
    int *m2; 
    m2 = (int *)malloc(size); 
    
    hipMemcpy(m2, dev_m, size, hipMemcpyDeviceToHost); 
    
    int ok=1; 
    for(int i=0;i<(width*width);++i) { 
        if(m[i]!=m2[i]) ok=0; 
    } 
    
    fprintf(stdout, "%s\n", ok?"ok":"error"); 
    
    free(m); 
    free(m2); 
    hipFree(m); 
    
    return 0; 
}


