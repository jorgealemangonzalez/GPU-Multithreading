
#include <hip/hip_runtime.h>
#include "stdio.h"
#define N 10

__global__ void add(int *a, int *b, int *c)
{
	int tid = threadIdx.x;
	c[tid]=a[tid]+b[tid];
}

int main()
{
	int a[N], b[N], c[N];//host 
	int *dev_a, *dev_b, *dev_c;//device

	hipMalloc((void**)&dev_a, N*sizeof(int) );  
	hipMalloc((void**)&dev_b, N*sizeof(int) );
	hipMalloc((void**)&dev_c, N*sizeof(int) );


	for (int i = 0; i < N; i++){
		a[i] = i,
		b[i] = 1;
	}

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice); //host to device
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	int blockSize = 1024;
	int gridSize = (int)ceil((int)N/blockSize);
	
	add<<<N,N>>>(dev_a,dev_b,dev_c); //Execute Kernel
	//Call CUDA kernel
	
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);//Copy memory from device to host
	//copy array to host
	for (int i = 0; i < N; i++)
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	
	hipFree(dev_a);//free device mem
	hipFree(dev_b);
	hipFree(dev_c);

	/*
	free(a);//free host
	free(b);
	free(c);*/
	return 0;

}
