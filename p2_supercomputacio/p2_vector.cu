#include "hip/hip_runtime.h"
#include "stdio.h"
#define N 10

__global__ void add(int *a, int *b, int *c)
{
	int tid;
	
}

int main()
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	hipMalloc(/*...*/);
	hipMalloc(/*...*/);
	hipMalloc(/*...*/);

	for (int i = 0; i < N; i++){
		a[i] = i,
		b[i] = 1;
	}

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	//Call CUDA kernel

	hipMemcpy();//Copy memory from device to host

	for (int i = 0; i < N; i++)
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	
	return 0;

}
