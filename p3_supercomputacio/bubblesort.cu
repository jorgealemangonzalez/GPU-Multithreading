
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define MIN(a,b) (a < b ? a : b)
#define PINT 1
static const int N = 50000;


__global__ void bubble_sort(int *array, int iteracio)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	/*
	for(int i =0 ; i < N - id ; ++i){ //usamos el id del array para saber hasta donde recorrer. Es esto lo k se debe hacer?
		if(array[i] > array[i+1]){
			int aux = array[i];
			array[i]=array[i+1];
			array[i+1] = aux;
		}
	}*/
	/*int aux;
	if(array[id-1] < array [id]){
		aux = array[id-1];
		array[id-1] = array[id];
		array[id] = aux;
	}*/
	if(iteracio%2 == 0 ){
		if(array[2*id] > array[2*id+1])
		{
			int aux = array[2*id];
			array[2*id] = array[2*id+1];
			array[2*id+1] = aux;
		}
	}else{
		if(array[2*id+1] > array[2*id+2]){
			int aux = array[2*id+1];
			array[2*id+1] = array[2*id+2];
			array[2*id+2] = aux;
		}
	}

}
//swap values at memory locations
void swap(int *elem1, int *elem2)//Exange the values stored in the memory spaces elem1 and elem2
{								 //To do it we create an auxiliar variable where the value of the first element
								 //is stored while we store the value of the second iin the first one
//Your code here
	int aux = *elem1; //We use an auxiliar variable to do the swap
	*elem1 = *elem2;
	*elem2 = aux;

}

//Bubble sort algorithm to sort arrays in ascending order
void bubbleSort(int * const array, const int size)
{


//Your code here

//1. Iterate along array elements

//2. swap adjacent elements if they are out of order

	int i , j;
	for(i = 0; i < size-1 ; ++i)
		for(j = 0 ; j < size - i -1 ; ++j)
			if(*(array + j) > *(array + j + 1))
				swap(array+j , array+j+1);


}

int main(int argc, char const *argv[]) 
{
	
    srand(time(NULL));


	int a[N];
	int *dev_a;



	for(int i=0;i<N;i++)
		a[i] = (int)rand()/(int)(RAND_MAX/300.0);
	

#if PRINT
	printf("desordenat\n");	
	for(int i=0;i<N;i++)
		printf("%d ", a[i]);
#endif

 	//execucio al CPU

 	clock_t t_host = clock();
 	bubbleSort(a,N);
 	t_host = clock() - t_host;
   	double time_taken_host = ((double)t_host)/CLOCKS_PER_SEC;
   	printf("CPU: %f segons\n",time_taken_host);

   	//execucio GPU

 	//device Memmory
	hipMalloc((void**)&dev_a, N*sizeof(int) );

	hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
	
	int threads_block = MIN(512,N);
	while(N%threads_block != 0)--threads_block;
	int blocks = N / threads_block;
	//execucio 

	clock_t t_device = clock();
	for (int it = 0; it <= 2*N; it++) {

		//Crida al kernel
		if(it%2 == 0){
			bubble_sort<<<1,(N/2)>>>(dev_a,it);
		}else{
			bubble_sort<<<1,(N/2)>>>(dev_a,it);
		}
			//nose el porque se debe hacer hasta 2*N
		
	}
	hipMemcpy(a,dev_a,N*sizeof(int),hipMemcpyDeviceToHost);

	t_device = clock() - t_device;
    double time_taken_device = ((double)t_device)/CLOCKS_PER_SEC; 
    printf("GPU %f segons \n", time_taken_device);
	

	printf("\nOrdenat\n");
	for(int i=0;i<N;i++)
		printf("%d ", a[i]);


	hipFree(dev_a);
	
	return 0;
}
