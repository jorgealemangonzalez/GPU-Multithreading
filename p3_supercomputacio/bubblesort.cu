
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define MIN(a,b) (a < b ? a : b)
static const int N = 50;


__global__ void bubble_sort(int *array, int iteracio)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	/*
	for(int i =0 ; i < N - id ; ++i){ //usamos el id del array para saber hasta donde recorrer. Es esto lo k se debe hacer?
		if(array[i] > array[i+1]){
			int aux = array[i];
			array[i]=array[i+1];
			array[i+1] = aux;
		}
	}*/
	int aux;
	if(array[id-1] < array [id]){
		aux = array[id-1];
		array[id-1] = array[id];
		array[id] = aux;
	}
}

int main(int argc, char const *argv[]) 
{
	
    srand(time(NULL));


	int a[N];
	int *dev_a;



	for(int i=0;i<N;i++)
		a[i] = (int)rand()/(int)(RAND_MAX/300.0);
	printf("desordenat\n");
	for(int i=0;i<N;i++)
		printf("%d ", a[i]);


 	
 	//device Memmory
	hipMalloc((void**)&dev_a, N*sizeof(int) );

	hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
	
	int threads_block = MIN(512,N);
	while(N%threads_block != 0)--threads_block;
	int blocks = N / threads_block;
	for (int it = 0; it <= 0; it++) {

		//Crida al kernel
		bubble_sort<<<blocks,threads_block>>>(dev_a,it);	//nose el porque se debe hacer hasta 2*N
		
	}

	hipMemcpy(a,dev_a,N*sizeof(int),hipMemcpyDeviceToHost);

	

	printf("\nOrdenat\n");
	for(int i=0;i<N;i++)
		printf("%d ", a[i]);



	
	return 0;
}
