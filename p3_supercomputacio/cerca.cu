#include "hip/hip_runtime.h"
#include<stdio.h>
#include <time.h>
#include<math.h>
#define TRUE 1
#define FALSE 0
#define MIN(a,b) (a < b?a:b )
static const int N = 15;

__global__ void cerca_array_device(int *array,int *valor,int *res)
{

    int id = threadIdx.x + blockIdx.x * blockDim.x;
	
	*res = (array[id] == *valor ? TRUE : FALSE);


}


__host__ bool cerca_array_host(int *array, int valor)
{

	for(int i = 0 ; i < N ; ++i)
		if(array[i] == valor)
			return true;
	return false;
}

int main()
{

    srand(time(NULL));

    int a[N],valor;

   

   
 for(int i=0;i<N;i++)
		a[i] = (int)rand()/(int)(RAND_MAX/300.0);

 for(int i=0;i<N;i++)
	printf("valor: %d \t", a[i]);

   
    printf("\nNombre a cercar: ");
    scanf("%d",&valor);


   //Execució a la CPU
    int res;
    clock_t t_host = clock();
    res = cerca_array_host(a,valor);
    t_host = clock() - t_host;
    double time_taken_host = ((double)t_host)/CLOCKS_PER_SEC;
 
    printf("CPU: %f segons \n", time_taken_host);
	
	if(res == TRUE)
		printf("host: We found the number\n");
	else
		printf("host: We don't found the number :(\n");
	
    clock_t t_device = clock();
    int *dev_array , *dev_value , *dev_res;
	hipMalloc((void**)&dev_array, N*sizeof(int) );
	hipMalloc((void**)&dev_value, sizeof(int) );
	hipMalloc((void**)&dev_res, sizeof(int) );
	
    int threads_block = MIN(512,N);
	while(N%threads_block != 0)--threads_block;
	int blocks = N / threads_block;
	
	cerca_array_device<<<blocks,threads_block>>>(dev_array,dev_value,dev_res); 
	hipMemcpy(res, dev_res, sizeof(int), hipMemcpyDeviceToHost);//Copy memory from device to host
    t_device = clock() - t_device;

    double time_taken_device = ((double)t_device)/CLOCKS_PER_SEC; 
        printf("GPU %f segons \n", time_taken_device);

   

	//Printa si ha cercat el nombre
	if(res == TRUE)
		printf("device: We found the number\n");
	else
		printf("device:  don't found the number :(\n");
   
return 0;


}

