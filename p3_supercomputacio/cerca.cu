
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <time.h>

static const int N = 15;

__global__ void cerca_array_device(int *array,int *valor,int *res)
{

    int id;

	
	//El vostre codi aquí




}


__host__ bool cerca_array_host(int *array, int valor)
{

	//El vostre codi aquí

}

int main()
{

    srand(time(NULL));

    int a[N],valor;

   

   
 for(int i=0;i<N;i++)
		a[i] = (int)rand()/(int)(RAND_MAX/300.0);

 for(int i=0;i<N;i++)
	printf("valor: %d \t", a[i]);

   
    printf("\nNombre a cercar: ");
    scanf("%d",&valor);


   //Execució a la CPU
    clock_t t_host = clock();
    cerca_array_host(a,valor);
    t_host = clock() - t_host;
    double time_taken_host = ((double)t_host)/CLOCKS_PER_SEC;
 
    printf("CPU: %f segons \n", time_taken_host);


    clock_t t_device = clock();
	//Crida al kernel---
    t_device = clock() - t_device;

    double time_taken_device = ((double)t_device)/CLOCKS_PER_SEC; 
        printf("GPU %f segons \n", time_taken_device);

   

	//Printa si ha cercat el nombre

   
return 0;


}

