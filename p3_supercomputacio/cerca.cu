
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <time.h>
#define TRUE 1
#define FALSE 0
#define MIN(a,b) (a < b?a:b )
static const int N = 150;

__global__ void cerca_array_device(int *array,int *valor,int *res)
{
	int b;
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(*res == FALSE && *valor == array[id]){
    	*res = TRUE;
    	/*for(int i = 0; i <= 1000000; i++) //demostrar que si se hacen mas operacines en la busqueda, los threads son
			b = (b*70)/3;*/				//muxo mas rapidos.
    }

}


__host__ bool cerca_array_host(int *array, int valor)
{
	int b;
	for(int i = 0 ; i < N ; ++i){
		if(array[i] == valor){
			return true;
		}
		/*for(int i = 0; i <= 1000000; i++)
			b = (b*70)/3;*/
	}

	return false;
}

int main()
{

    srand(time(NULL));

    int a[N],valor;

   

   
 for(int i=0;i<N;i++)
		a[i] = (int)rand()/(int)(RAND_MAX/300.0);

 for(int i=0;i<N;i++)
	printf("valor: %d \t", a[i]);

   
    printf("\nNombre a cercar: ");
    scanf("%d",&valor);


   //Execució a la CPU
    int res;
    clock_t t_host = clock();
    res = cerca_array_host(a,valor);
    t_host = clock() - t_host;
    double time_taken_host = ((double)t_host)/CLOCKS_PER_SEC;
 
    printf("CPU: %f segons \n", time_taken_host);
	
	if(res == TRUE)
		printf("host: We found the number\n");
	else
		printf("host: We don't found the number :(\n");
	
   
    
    int *dev_array , *dev_value , *dev_res;
	hipMalloc((void**)&dev_array, N*sizeof(int) );
	hipMalloc((void**)&dev_value, sizeof(int) );
	hipMalloc((void**)&dev_res, sizeof(int) );
	
	res = FALSE;
	hipMemcpy(dev_array, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_value, &valor, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_res, &res, sizeof(int), hipMemcpyHostToDevice);
	
    int threads_block = MIN(512,N);
	while(N%threads_block != 0)--threads_block;
	int blocks = N / threads_block;
	
	clock_t t_device = clock();
	cerca_array_device<<<blocks,threads_block>>>(dev_array,dev_value,dev_res); 
	hipMemcpy(&res, dev_res, sizeof(int), hipMemcpyDeviceToHost);//Copy memory from device to host
    t_device = clock() - t_device;

    double time_taken_device = ((double)t_device)/CLOCKS_PER_SEC; 
        printf("GPU %f segons \n", time_taken_device);

   	hipFree(dev_array);//free device mem
	hipFree(dev_value);
	hipFree(dev_res);


	//Printa si ha cercat el nombre
	if(res == TRUE)
		printf("device: We found the number\n");
	else
		printf("device: We don't found the number :(\n");
		
   
return 0;


}

