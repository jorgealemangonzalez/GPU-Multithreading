#include <stdio.h> 
#include <assert.h> 
#include <stdlib.h> 
#include <stdio.h> 
#include <hip/hip_runtime.h> 

struct BITMAPFILEHEADER 
{ 
    char bfType[2]; 
    int bfSize; 
    int bfReserved; 
    int bfOffBits; 
}; 

struct BITMAPINFOHEADER { 
    int biSize; 
    int biWidth; 
    int biHeight; 
    short biPlanes; 
    short biBitCount; 
    int biCompression; 
    int biSizeImage; 
    int biXPelsPerMeter; 
    int biYPelsPerMeter; 
    int biClrUsed; 
    int biClrImportant; 
}; 

int write_bitmap(const char *filename, int width, int height, char *red, char *green, char *blue) 
{ 

	int bytes_per_line = (3 * (width + 1) / 4) * 4; 
	unsigned char *image_line = (unsigned char *)malloc(bytes_per_line); 

struct BITMAPFILEHEADER bmph; 
    bmph.bfType[0] = 'B'; 
    bmph.bfType[1] = 'M'; 
    bmph.bfReserved = 0; 
    bmph.bfOffBits = 54; 
    bmph.bfSize = bmph.bfOffBits + bytes_per_line * height; 

    struct BITMAPINFOHEADER bmih; 
    bmih.biSize = 40; 
    bmih.biWidth = width; 
    bmih.biHeight = height; 
    bmih.biPlanes = 1; 
    bmih.biBitCount = 24; 
    bmih.biCompression = 0; 
    bmih.biSizeImage = bytes_per_line * height; 
    bmih.biXPelsPerMeter = 0; 
    bmih.biYPelsPerMeter = 0; 

 bmih.biClrUsed = 0; 
    bmih.biClrImportant = 0; 
    FILE *fit; 
    if((fit = fopen (filename, "wb"))==0) { 
        free(image_line);
	return -1;
    } 

fwrite(&bmph.bfType, 2, 1, fit); 
    fwrite(&bmph.bfSize, 4, 1, fit); 
    fwrite(&bmph.bfReserved, 4, 1, fit); 
    fwrite(&bmph.bfOffBits, 4, 1, fit); 
    
    fwrite(&bmih.biSize, 4, 1, fit); 
    fwrite(&bmih.biWidth, 4, 1, fit); 
    fwrite(&bmih.biHeight, 4, 1, fit); 
    fwrite(&bmih.biPlanes, 2, 1, fit); 
    fwrite(&bmih.biBitCount, 2, 1, fit); 
    fwrite(&bmih.biCompression, 4, 1, fit); 
    fwrite(&bmih.biSizeImage, 4, 1, fit); 
    fwrite(&bmih.biXPelsPerMeter, 4, 1, fit); 
    fwrite(&bmih.biYPelsPerMeter, 4, 1, fit); 
    fwrite(&bmih.biClrUsed, 4, 1, fit); 
    fwrite(&bmih.biClrImportant, 4, 1, fit);

	for(int i=height-1;i>=0;i--) {
        for (int j=0;j<width;j++) { 
            int pos = (width * i + j); 
            image_line[3*j] = blue[pos]; 
            image_line[3*j+1] = green[pos]; 
            image_line[3*j+2] = red[pos]; 
        } 
        fwrite((void *)image_line, bytes_per_line, 1, fit); 
    } 
    free(image_line); 
    fclose(fit); 

return 0; 

} 

void mandel_host(char *red, char *green, char *blue, int width, int height) 
{ 
    for(int pos_x=0;pos_x<width;pos_x++) { 
        for(int pos_y=0;pos_y<height;pos_y++) { 
	    float x0 = ((float)pos_x)*3.5/((float)width)-2.5;
	    float y0 = ((float)pos_y)*2.0/((float)height)-1.0;
            float x = 0.0; 
            float y = 0.0; 
            int iteration = 0; 
            int max_iteration = 256; 
            while(x*x + y*y <= 4 && iteration < max_iteration) { 
		float xtemp = x*x - y*y + x0;
                y = 2*x*y + y0; 
                x = xtemp; 
                iteration++; 
            } 
            int index = width*pos_y + pos_x; 
  
            if(iteration==max_iteration) { 
                iteration = 0; 
            } 
            red[index] = iteration; 
            green[index] = iteration; 
            blue[index] = iteration; 
        } 
    } 
}

__global__ void mandel_cuda(char *red, char *green, char *blue, int width, int height) 
{

	/* kernel que calcula un pixel */ 
    	/* Per saber quin pixel és, cal tenir en compte totes les dimensions 
       	del grid (el número de blocs i el número de threads */ 
    	/* Podeu fer servir els valors de 
       		blockIdx.x, blockIdx.y 
       		gridDim.x, gridDim.y 
       		threadIdx.x, threadIdx.y 
       		blockDim.x, blockDim.y */ 


}

void fes_host(int width, int height) 
{ 
    size_t buffer_size = sizeof(char) * width * height; 
    char *image_red = (char *)malloc(buffer_size); 
    char *image_green = (char *)malloc(buffer_size); 
    char *image_blue = (char *)malloc(buffer_size); 
    mandel_host(image_red, image_green, image_blue, width, height); 
    // Now write the file 
    write_bitmap("output_host.bmp", width, height, image_red, 
                    image_green, image_blue); 
    free(image_red); 
    free(image_green); 
    free(image_blue); 
} 

void fes_cuda(int width, int height) 
{ 
    size_t buffer_size = sizeof(char) * width * height; 
    char *image_red; 
    char *image_green; 
    char *image_blue; 

/* cal reservar la memòria del dispositiu */ 
    dim3 blockDim(16, 16, 1); 
    dim3 gridDim(width / blockDim.x, height / blockDim.y, 1); 
    
    mandel_cuda<<< gridDim, blockDim, 0 >>>(image_red, image_green, image_blue, width, height); 
    char *host_image_red; 
    char *host_image_green; 
    char *host_image_blue; 
    
    /* cal copiar els valors de la imatge al host */ 
    // Now write the file 
    write_bitmap("output_cuda.bmp", width, height, host_image_red, 
                    host_image_green, host_image_blue); 
    /* cal alliberar la memòria del dispositiu i del host */ 
} 
int main(int argc, const char * argv[]) { 
    fes_cuda(5120, 5120); 
    fes_host(5120, 5120); 
    return 0; 
} 






