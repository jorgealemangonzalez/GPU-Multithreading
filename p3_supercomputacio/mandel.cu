#include <stdio.h> 
#include <assert.h> 
#include <stdlib.h> 
#include <stdio.h> 
#include <hip/hip_runtime.h> 
#include <time.h>


struct BITMAPFILEHEADER 
{ 
    char bfType[2]; 
    int bfSize; 
    int bfReserved; 
    int bfOffBits; 
}; 

struct BITMAPINFOHEADER { 
    int biSize; 
    int biWidth; 
    int biHeight; 
    short biPlanes; 
    short biBitCount; 
    int biCompression; 
    int biSizeImage; 
    int biXPelsPerMeter; 
    int biYPelsPerMeter; 
    int biClrUsed; 
    int biClrImportant; 
}; 

int write_bitmap(const char *filename, int width, int height, char *red, char *green, char *blue) 
{ 

	int bytes_per_line = (3 * (width + 1) / 4) * 4; 
	unsigned char *image_line = (unsigned char *)malloc(bytes_per_line); 

struct BITMAPFILEHEADER bmph; 
    bmph.bfType[0] = 'B'; 
    bmph.bfType[1] = 'M'; 
    bmph.bfReserved = 0; 
    bmph.bfOffBits = 54; 
    bmph.bfSize = bmph.bfOffBits + bytes_per_line * height; 

    struct BITMAPINFOHEADER bmih; 
    bmih.biSize = 40; 
    bmih.biWidth = width; 
    bmih.biHeight = height; 
    bmih.biPlanes = 1; 
    bmih.biBitCount = 24; 
    bmih.biCompression = 0; 
    bmih.biSizeImage = bytes_per_line * height; 
    bmih.biXPelsPerMeter = 0; 
    bmih.biYPelsPerMeter = 0; 

 bmih.biClrUsed = 0; 
    bmih.biClrImportant = 0; 
    FILE *fit; 
    if((fit = fopen (filename, "wb"))==0) { 
        free(image_line);
	return -1;
    } 

fwrite(&bmph.bfType, 2, 1, fit); 
    fwrite(&bmph.bfSize, 4, 1, fit); 
    fwrite(&bmph.bfReserved, 4, 1, fit); 
    fwrite(&bmph.bfOffBits, 4, 1, fit); 
    
    fwrite(&bmih.biSize, 4, 1, fit); 
    fwrite(&bmih.biWidth, 4, 1, fit); 
    fwrite(&bmih.biHeight, 4, 1, fit); 
    fwrite(&bmih.biPlanes, 2, 1, fit); 
    fwrite(&bmih.biBitCount, 2, 1, fit); 
    fwrite(&bmih.biCompression, 4, 1, fit); 
    fwrite(&bmih.biSizeImage, 4, 1, fit); 
    fwrite(&bmih.biXPelsPerMeter, 4, 1, fit); 
    fwrite(&bmih.biYPelsPerMeter, 4, 1, fit); 
    fwrite(&bmih.biClrUsed, 4, 1, fit); 
    fwrite(&bmih.biClrImportant, 4, 1, fit);

	for(int i=height-1;i>=0;i--) {
        for (int j=0;j<width;j++) { 
            int pos = (width * i + j); 
            image_line[3*j] = blue[pos]; 
            image_line[3*j+1] = green[pos]; 
            image_line[3*j+2] = red[pos]; 
        } 
        fwrite((void *)image_line, bytes_per_line, 1, fit); 
    } 
    free(image_line); 
    fclose(fit); 

return 0; 

} 

void mandel_host(char *red, char *green, char *blue, int width, int height) 
{ 
    for(int pos_x=0;pos_x<width;pos_x++) { 
        for(int pos_y=0;pos_y<height;pos_y++) { 
	    float x0 = ((float)pos_x)*3.5/((float)width)-2.5;
	    float y0 = ((float)pos_y)*2.0/((float)height)-1.0;
            float x = 0.0; 
            float y = 0.0; 
            int iteration = 0; 
            int max_iteration = 256; 
            while(x*x + y*y <= 4 && iteration < max_iteration) { 
		float xtemp = x*x - y*y + x0;
                y = 2*x*y + y0; 
                x = xtemp; 
                iteration++; 
            } 
            int index = width*pos_y + pos_x; 
  
            if(iteration==max_iteration) { 
                iteration = 0; 
            } 
            red[index] = iteration; 
            green[index] = iteration; 
            blue[index] = iteration; 
        } 
    } 
}

__global__ void mandel_cuda(char *red, char *green, char *blue, int width, int height) 
{

	/* kernel que calcula un pixel */ 
    	/* Per saber quin pixel és, cal tenir en compte totes les dimensions 
       	del grid (el número de blocs i el número de threads */ 
    	/* Podeu fer servir els valors de 
       		blockIdx.x, blockIdx.y 
       		gridDim.x, gridDim.y 
       		threadIdx.x, threadIdx.y 
       		blockDim.x, blockDim.y */ 
            
    int pos_x = threadIdx.x+blockDim.x*blockIdx.x;
    int pos_y = threadIdx.y+blockDim.y*blockIdx.y;

    float x0 = ((float)pos_x)*3.5/((float)width)-2.5;
    float y0 = ((float)pos_y)*2.0/((float)height)-1.0;
        float x = 0.0; 
        float y = 0.0; 
        int iteration = 0; 
        int max_iteration = 256; 
        while(x*x + y*y <= 4 && iteration < max_iteration) { 
    float xtemp = x*x - y*y + x0;
            y = 2*x*y + y0; 
            x = xtemp; 
            iteration++; 
        } 
        int index = width*pos_y + pos_x; 

        if(iteration==max_iteration) { 
            iteration = 0; 
        } 
        red[index] = iteration; 
        green[index] = iteration; 
        blue[index] = iteration;

}

void fes_host(int width, int height) 
{ 
    size_t buffer_size = sizeof(char) * width * height; 
    char *image_red = (char *)malloc(buffer_size); 
    char *image_green = (char *)malloc(buffer_size); 
    char *image_blue = (char *)malloc(buffer_size); 
    mandel_host(image_red, image_green, image_blue, width, height); 
    // Now write the file 
    write_bitmap("output_host.bmp", width, height, image_red, 
                    image_green, image_blue); 
    free(image_red); 
    free(image_green); 
    free(image_blue); 
} 

void fes_cuda(int width, int height) 
{ 
    size_t buffer_size = sizeof(char) * width * height; 
    char *image_red; 
    char *image_green; 
    char *image_blue; 
 
    hipMalloc((void**)&image_red, buffer_size);
    hipMalloc((void**)&image_green, buffer_size);
    hipMalloc((void**)&image_blue, buffer_size);

    clock_t t_device = clock();

    dim3 blockDim(6, 6,1); 
    dim3 gridDim(width / blockDim.x, height / blockDim.y,1); 
    
    mandel_cuda<<< gridDim, blockDim,0>>>(image_red, image_green, image_blue, width, height); 
    char *host_image_red = (char*)malloc(buffer_size); 
    char *host_image_green= (char*)malloc(buffer_size); 
    char *host_image_blue= (char*)malloc(buffer_size); 
    
    /* cal copiar els valors de la imatge al host */ 

    hipMemcpy(host_image_red,image_red,buffer_size,hipMemcpyDeviceToHost);
    hipMemcpy(host_image_green,image_green,buffer_size,hipMemcpyDeviceToHost);
    hipMemcpy(host_image_blue,image_blue,buffer_size,hipMemcpyDeviceToHost);
    // Now write the file 
    write_bitmap("output_cuda.bmp", width, height, host_image_red, 
                    host_image_green, host_image_blue); 
    t_device = clock() - t_device;
    double time_taken_device = ((double)t_device)/CLOCKS_PER_SEC; 
    printf("GPU %f segons with %d threats \n", time_taken_device,blockDim.x);
    /* cal alliberar la memòria del dispositiu i del host */ 
    hipFree(image_blue);
    hipFree(image_green);
    hipFree(image_red);
    free(host_image_blue);
    free(host_image_green);
    free(host_image_red);
} 

unsigned char* readBMP(const char* filename)
{
    int i;
    FILE* f = fopen(filename, "rb");
    unsigned char info[54];
    fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

    // extract image height and width from header
    int width = *(int*)&info[18];
    int height = *(int*)&info[22];

    int size = 3 * width * height;
    unsigned char* data = new unsigned char[size]; // allocate 3 bytes per pixel
    fread(data, sizeof(unsigned char), size, f); // read the rest of the data at once
    fclose(f);

    for(i = 0; i < size; i += 3)
    {
            unsigned char tmp = data[i];
            data[i] = data[i+2];
            data[i+2] = tmp;
    }

    return data;
}

int main(int argc, const char * argv[]) { 
    fes_cuda(5120, 5120); 
    fes_host(5120, 5120); 
    unsigned char *c , *h;
    c = readBMP("output_cuda.bmp");
    h = readBMP("output_host.bmp");
    int errors =0;
    int lengc = 5120*5120;
    for(int i = 0 ; i < lengc;++i){
        if(c[i] != h[i]){
            errors++;
        }
    }
    if(errors)printf("There are no difference,have %d errors\n",errors);
    else printf("There are no difference\n");
    return 0; 
} 






