
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

static const int N = 16;
//static const int N = 32;
//static const inst N = 13;
//...


//Kernel que distribueix la l'execució a la grid
__global__ void organitza_grid(int *array) {

    int idx_x;
    int idx_y; 

    //Distribueix la grid(blocks i threads) com a un array unidimensional i calcula l'index d'aquesta distribució. 
    //On cada index correspon a un thread de la grid
    
	//....
    
     //Recupera l'index del block a la grid

	//...



    //Guarda resultad al array

	//...

}


__host__ void printa(int *array)
{

//Els vostre codi...


}


int main(void) {



    int *dev_a;
    int *array;

    // Reserva memoria al host i al device


    //Crea blocks de dos dimensions amb diferent nombre de threads. Ex: Comença amb 4x4
    dim3 block_dim;
	//...

    // Crea i inicialitza una grid en 2 dimensions
    dim3 grid_dim;
	//...


    organitza_grid<<<grid_dim, block_dim>>>(dev_a);



    // Printa els resultats de l'organització de la grid
    printa(array);
   



    return 0;
}
