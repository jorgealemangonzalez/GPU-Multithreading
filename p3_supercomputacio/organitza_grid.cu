
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

static const int N = 16;
//static const int N = 32;
//static const inst N = 13;
//...


//Kernel que distribueix la l'execució a la grid
__global__ void organitza_grid(int *array) {



    //Distribueix la grid(blocks i threads) com a un array unidimensional i calcula l'index d'aquesta distribució. 
    //On cada index correspon a un thread de la grid
    int idx = threadIdx.x;
    int idy = threadIdx.y;
    int idblocy = blockIdx.y;
    int idblocx = blockIdx.x;
    int width = gridDim.x * blockDim.x;
    int id_array = idy*width + idx + idblocx * blockDim.x + idblocy * width * blockDim.y;
    //array [id_array] = (2*idblocy)+idblocx;
    //array[idblocy*width+idblocx*blockDim.x+idy*blockDim.y+idx] = idblocy*width+idblocx*blockDim.x+idy*blockDim.y+idx;
	
    //....
    
     //Recupera l'index del block a la grid

	//...



    //Guarda resultad al array

	//...

}


__host__ void printa(int *array)
{

//Els vostre codi...
    for(int i = 0 ; i < N ; ++i){
        printf("%d-",array[i]);
    }

}


int main(void) {


    //blockDim.x -- number threads in block
    //blockid -- block index
    //gridim number blocks in grid


    int *dev_a;
    int *array;
    int size = N*sizeof(int);

    // Reserva memoria al host i al device
    array = (int *)malloc(size); 

    hipMalloc((void **)&dev_a, size); 



    //Crea blocks de dos dimensions amb diferent nombre de threads. Ex: Comença amb 4x4
    dim3 block_dim(4,4); //4 threads x bloque
	//...

    // Crea i inicialitza una grid en 2 dimensions
    //dim3 grid_dim(sqrt(size)/block_dim.x, sqrt(size)/block_dim.y);
    dim3 grid_dim(2,2);
	//...

    hipMemset(dev_a,0,N);
    organitza_grid<<<grid_dim, block_dim>>>(dev_a);
    hipMemcpy(array,dev_a,N,hipMemcpyDeviceToHost);


    // Printa els resultats de l'organització de la grid
    printa(array);
   



    return 0;
}
