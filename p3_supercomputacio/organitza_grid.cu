
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DEBUGG 1

//static const int N = 16;
static const int N = 32;
//static const int N = 13;
//...


//Kernel que distribueix la l'execució a la grid
__global__ void organitza_grid(int *array) {



    //Distribueix la grid(blocks i threads) com a un array unidimensional i calcula l'index d'aquesta distribució. 
    //On cada index correspon a un thread de la grid

    int idx = threadIdx.x;
    int idy = threadIdx.y;
    int idblocy = blockIdx.y;
    int idblocx = blockIdx.x;
    int width = gridDim.x * blockDim.x;
    int id_array = (idy*width + idx) + (idblocx * blockDim.x) + (idblocy * width * blockDim.y); 
    array[id_array]=(2*idblocy)+idblocx;
    
    //....
     //Recupera l'index del block a la grid
    //...

    //Guarda resultad al array

    //...

}


__host__ void printa(int *array,int sizex,int sizey)
{

//Els vostre codi...
    for(int i = 0 ; i < sizey ; ++i){      //impresion de la grid dependiendo del tamaño en el eje x e y
        for(int j = 0 ; j < sizex; ++j){
            printf("%d ",array[i*sizex+j]);
        }
        printf("\n");
    }
}   



int main(void) {


    //blockDim.x -- number threads in block
    //blockid -- block index
    //gridim number blocks in grid


    int *dev_a , numbloq , gridsizex,gridsizey;
    int *array;
    int size = N*sizeof(int);

    // Reserva memoria al host i al device
    array = (int *)malloc(size); 

    hipMalloc((void **)&dev_a, size); 

    memset(array,0,N); //inicializamos en 0 el array

    hipMemcpy(dev_a,array,size,hipMemcpyHostToDevice); //copiamos el array del host al device

    //Crea blocks de dos dimensions amb diferent nombre de threads. Ex: Comença amb 4x4
    dim3 block_dim(2,2); //4 threads x bloque, dimension 2*2
    //...

    numbloq = N/(block_dim.y*block_dim.x); //numero de bloques que tendremos

    // Crea i inicialitza una grid en 2 dimensions
    dim3 grid_dim(block_dim.x, numbloq/block_dim.y);  //la grid siempre tendra dos bloques en el eje x

    gridsizex = grid_dim.x*block_dim.x;
    gridsizey = grid_dim.y*block_dim.y;
    //...
#if DEBUGG
    printf("Number blocks %d\n",numbloq);
    printf("Dim block (x,y) %d-%d",block_dim.x,block_dim.y);
    printf("\nDim Grid (blocks)(x,y) %d-%d",grid_dim.x,grid_dim.y);
    printf("\ngrid size (threads)(x,y) %d-%d\n",gridsizex,gridsizey);
#endif

    organitza_grid<<<grid_dim, block_dim>>>(dev_a);
    hipMemcpy(array,dev_a,size,hipMemcpyDeviceToHost);


    // Printa els resultats de l'organització de la grid
    printa(array,gridsizex,gridsizey);
   



    return 0;
}
