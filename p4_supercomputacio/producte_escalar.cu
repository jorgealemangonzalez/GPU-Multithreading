#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define imin(a,b) (a<b?a:b)

//Nombre de posicions del vector
const int N = 33 * 1024;

//Nombre de threads per cada block
const int threadsPerBlock = 256;

//Escollim el nombre de blocs a la grid
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1)/threadsPerBlock);

__global__ void producte_escalar(float* a, float* b, float* c) {

	//Declarem un vector cache de memoria compartida
	__shared__ float vectorShared[N];
	//Inicialitzem l'index amb una combinació de blocks i threads
	int tid = hreadIdx.x + blockIdx.x * blockDim.x;

	//Inicialitzem l'index del cache. PISTA: Es compartid per cada bloc, és a dir, cada block te una copia
	int indexCache = 
	
	float temp = 0;
	//Fem la suma de productes al block actual
	while(tid < N) {
		

		tid += blockDim.x * gridDim.x;
	}

	//Emmagatzem el valor temporal de la suma de productes a la cache
	

	//ara hem de llegir els valors de la cahche, però abans ens hem d'assegurar que els valors s'han escrit


	//Reduim el vector
	int i = blockDim.x/2;
	while(i != 0) {
		if(indexCache < i) {
			cache[indexCache] += cache[indexCache + i];
		}

		__syncthreads();
		i /= 2;
	}

	//Retornem un vector c que tindrà tantes posicions com blocks estem fent servir.
	//Guardem a c el contingut de cache a la unica posició restant. Per què?
	c[/*...*/] =


}

int main()
{
	float *a, *b, *c;
	float *dev_a, *dev_b, *dev_c;
	int size = N*sizeof(float);

	//Reserva memoria a la CPU
	a = (float *)malloc(size);
	b = (float *)malloc(size);
	c = (float *)malloc(size);

	//Reserva memoria a la GPU.

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);



	//Emplenem els vector a i b
	for(int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = 2 * i;
	}

	//Copiem els arrays a i b a la GPU
	hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size,hipMemcpyHostToDevice);




	producte_escalar<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c);

	//Copiem l'array dev_c a c

	hipMemcpy(c,dev_c,size,hipMemcpyDeviceToHost);

	//Acabem de realitzar la suma de productes
	float value = 0;



printf("Producte escalar es: %f", value);

	//Allibera memoria a la GPU


	//Allibera memoria a la CPU


    return 0;
}

