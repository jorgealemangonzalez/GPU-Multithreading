
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define imin(a,b) (a<b?a:b)

//Nombre de posicions del vector
const int N = 33 * 1024;

//Nombre de threads per cada block
const int threadsPerBlock = 256;

//Escollim el nombre de blocs a la grid
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1)/threadsPerBlock);

__global__ void producte_escalar(float* a, float* b, float* c) {
	
	//Declarem un vector cache de memoria compartida
	__shared__ float cache[threadsPerBlock];
	//Inicialitzem l'index amb una combinació de blocks i threads
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	//Inicialitzem l'index del cache. PISTA: Es compartid per cada bloc, és a dir, cada block te una copia
	int indexCache = threadIdx.x;
	
	float temp = 0;
	//Fem la suma de productes al block actual
	while(tid < N) {
		temp += a[tid]*b[tid];
		
		tid += blockDim.x * gridDim.x;
	}
	
	//Emmagatzem el valor temporal de la suma de productes a la cache
	cache[indexCache] = temp;

	//ara hem de llegir els valors de la cache, però abans ens hem d'assegurar que els valors s'han escrit
	
	__syncthreads();
	
	//Reduim el vector
	int i = blockDim.x/2;
	while(i != 0) {
		if(indexCache < i) {
			cache[indexCache] += cache[indexCache + i];
		}

		__syncthreads();
		i /= 2;
	}
	
	//Retornem un vector c que tindrà tantes posicions com blocks estem fent servir.
	//Guardem a c el contingut de cache a la unica posició restant. Per què?------------------------> Porque n es potencia de dos
	c[blockIdx.x] = cache[0] ;

}

int main()
{
	float *a, *b, *c;
	float *dev_a, *dev_b, *dev_c;
	int size = N*sizeof(float);

	//Reserva memoria a la CPU
	a = (float *)malloc(size);
	b = (float *)malloc(size);
	c = (float *)malloc(blocksPerGrid * sizeof(float));

	//Reserva memoria a la GPU.

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, blocksPerGrid * sizeof(float));

	printf("Blocks : %d ",blocksPerGrid);

	//Emplenem els vector a i b
	for(int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = 2 * i;
	}

	//Copiem els arrays a i b a la GPU
	hipMemcpy(dev_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,size,hipMemcpyHostToDevice);




	producte_escalar<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c);

	//Copiem l'array dev_c a c

	hipMemcpy(c,dev_c,blocksPerGrid * sizeof(float),hipMemcpyDeviceToHost);

	//Acabem de realitzar la suma de productes
	float value = 0;

	for(int i = 0 ; i < blocksPerGrid ; ++i)
		value += c[i];
	float value_host = 0;
	for(int i = 0 ; i < N ; ++i){
		value_host += a[i]*b[i];
	}
	printf("Producte escalar es: %f pero deberia ser %f diff: %f\nThis is due to we are running less threads that we need\n", value , value_host ,value_host - value);

	//Allibera memoria a la GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	//Allibera memoria a la CPU
	free(a);
	free(b);
	free(c);
    return 0;
}

